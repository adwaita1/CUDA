#include "hip/hip_runtime.h"

*****************
Problem Statement
*****************
Program to Add two numbers on GPU.

****************************************************************************************/

#include<stdio.h>
#include<conio.h>

__global__ void add_number(float *ad,float *bd)
{

        *ad += *bd;                             //adding values in GPU memory
}
int main()
{
	float *a,*b;
	float *ad,*bd;
	size_t size = sizeof(float);

    //allocate memory on host
	a=(float*)malloc(size);
	b=(float*)malloc(size);
	
    //allocate memory on device
	hipMalloc(&ad,size);
	//printf("\nAfter hipMalloc for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMalloc(&bd,size);
	//printf("\nAfter hipMalloc for bd\n%s\n",hipGetErrorString(hipGetLastError()));
	
    printf("\nEnter two numbers\n");
    scanf("%f%f",a,b);

	//copy data from host memory to device memory
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for bd\n%s\n",hipGetErrorString(hipGetLastError()));
	
    //GPU timer code
    float time;
    hipEvent_t start,stop;			
	hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start,0);
	
    //launch kernel with only one thread
    add_number<<< 1,1 >>>(ad,bd);
	
    hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);			//time taken in kernel call calculated
	hipEventDestroy(start);
	hipEventDestroy(stop);

    //copy back the results
	hipMemcpy(a,ad,size,hipMemcpyDeviceToHost);
	//printf("\nAfter DeviceToHost Memcpy for a\n%s\n",hipGetErrorString(hipGetLastError()));
	
	//print the results
	printf("\nAddition of above two numbers on GPU evaluates to = %f",*a);
    printf("\n\nTime taken is %f (ms)\n",time);
    
    //deallocate host and device memories
    hipFree(ad); hipFree(bd);
	free(a);free(b);

	_getch();
    return 1;
}