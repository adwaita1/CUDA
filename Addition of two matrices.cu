#include "hip/hip_runtime.h"
#include<stdio.h>
#include<conio.h>

__global__ void add_matrices(float *ad,float *bd,float *cd,int N)
{
        cd[threadIdx.y * N + threadIdx.x] = ad[threadIdx.y * N + threadIdx.x] + bd[threadIdx.y * N + threadIdx.x];
}
int main()
{
	int N = 20;				//size of square matrix
    unsigned int i,j;
	
	float *a,*b;
	float *ad,*bd,*cd,*c;
	size_t size=sizeof(float)* N * N;

    //allocate memory on host
	a=(float*)malloc(size);
	b=(float*)malloc(size);
    c=(float*)malloc(size);
	
    //allocate memory on device
	hipMalloc(&ad,size);
	//printf("\nAfter hipMalloc for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMalloc(&bd,size);
	//printf("\nAfter hipMalloc for bd\n%s\n",hipGetErrorString(hipGetLastError()));
    hipMalloc(&cd,size);
	//printf("\nAfter hipMalloc for cd\n%s\n",hipGetErrorString(hipGetLastError()));
	
    //initialize host memory with its own indices
    for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
        {
			a[i * N + j]=(float)(i * N + j);
            b[i * N + j]= -(float)(i * N + j);
        }
    }

	//copy data from host memory to device memory
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for bd\n%s\n",hipGetErrorString(hipGetLastError()));

	//calculate execution configuration
    dim3 blocksize (N, N);		//each block contains N * N threads, each thread calculates 1 data element
    
    //GPU timer code
    float time;
    hipEvent_t start,stop;			
	hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start,0);
	
    add_matrices<<< 1, blocksize >>>(ad, bd, cd, N);
	
    hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);			//time taken in kernel call calculated
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);
	//printf("\nAfter DeviceToHost Memcpy for c \n%s\n",hipGetErrorString(hipGetLastError()));
	
	printf("Matrix A was---\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",a[i*N+j]);
		printf("\n");
	}
	printf("\nMatrix B was---\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",b[i*N+j]);
		printf("\n");
	}
	printf("\nAddition of A and B gives C----\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",c[i*N+j]);              //if correctly evaluated, all values will be 0
		printf("\n");
	}

	printf("\n\nTime taken is %f (ms)\n",time);
     
    //deallocate host and device memories
    hipFree(ad); hipFree(bd); hipFree (cd);
	free(a);free(b);free(c);

	_getch();
    return 1;
}