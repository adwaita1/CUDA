#include "hip/hip_runtime.h"
*****************
Problem Statement
*****************
Program to Add two Matrices using shared memory.

****************************************************************************************/

#include<stdio.h>
#include<conio.h>

//size of square matrix
#define N 20

__global__ void add_matrices(float *ad,float *bd, float *cd)
{
    //allocate shared memory for the two matrices
    __shared__ float ads [N][N];
    __shared__ float bds [N][N];

    //copy values from global memory into shared memory
    ads[threadIdx.y][threadIdx.x] = ad[threadIdx.y * N + threadIdx.x];
    bds[threadIdx.y][threadIdx.x] = bd[threadIdx.y * N + threadIdx.x];

    cd[threadIdx.y * N + threadIdx.x] = ads[threadIdx.y][threadIdx.x] + bds[threadIdx.y][threadIdx.x];
}
int main()
{
	unsigned int i,j;
	float *a,*b;
	float *ad,*bd,*cd,*c;
	size_t size=sizeof(float)* N * N;

    //allocate memory on host
	a=(float*)malloc(size);
	b=(float*)malloc(size);
    c=(float*)malloc(size);
	
    //allocate memory on device
	hipMalloc(&ad,size);
	//printf("\nAfter hipMalloc for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMalloc(&bd,size);
	//printf("\nAfter hipMalloc for bd\n%s\n",hipGetErrorString(hipGetLastError()));
    hipMalloc(&cd,size);
	//printf("\nAfter hipMalloc for cd\n%s\n",hipGetErrorString(hipGetLastError()));
	
    //initialize host memory with its own indices
    for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
        {
			a[i * N + j]=(float)(i * N + j);
            b[i * N + j]= -(float)(i * N + j);
        }
    }

	//copy data from host memory to device memory
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for bd\n%s\n",hipGetErrorString(hipGetLastError()));

	//calculate execution configuration
    dim3 blocksize (N, N);		//each block contains N * N threads, each thread calculates 1 data element
    
    //GPU timer code
    float time;
    hipEvent_t start,stop;			
	hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start,0);
	
    add_matrices<<< 1, blocksize, 2 * size >>>(ad, bd, cd);
	
    hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);			//time taken in kernel call calculated
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);
	//printf("\nAfter DeviceToHost Memcpy for c \n%s\n",hipGetErrorString(hipGetLastError()));
	
	printf("Matrix A was---\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",a[i*N+j]);
		printf("\n");
	}
	printf("\nMatrix B was---\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",b[i*N+j]);
		printf("\n");
	}
	printf("\nAddition of A and B gives C----\n");
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
			printf("%f ",c[i*N+j]);              //if correctly evaluated, all values will be 0
		printf("\n");
	}

	printf("\n\nTime taken is %f (ms)\n",time);
     
    //deallocate host and device memories
    hipFree(ad); hipFree(bd); hipFree (cd);
	free(a);free(b);free(c);

	_getch();
    return 1;
}