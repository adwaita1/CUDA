#include "hip/hip_runtime.h"
*****************
Problem Statement
*****************
Program to Add two vectors on GPU.

****************************************************************************************/

#include<stdio.h>
#include<conio.h>

__global__ void add_vectors(float *ad, float *bd, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
        ad[index] += bd[index];                             //adding values in GPU memory
}
int main()
{
	float *a,*b;
	float *ad,*bd;
    int N = 1024;                                  //size of vector to be added 
    unsigned int i, No_of_blocks, No_of_threads;
	size_t size = sizeof(float) * N;

    //allocate memory on host
	a=(float*)malloc(size);
	b=(float*)malloc(size);
	
    //allocate memory on device
	hipMalloc(&ad,size);
	//printf("\nAfter hipMalloc for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMalloc(&bd,size);
	//printf("\nAfter hipMalloc for bd\n%s\n",hipGetErrorString(hipGetLastError()));
	
    //initialize host memory with its own indices
    for(i=0; i<N; i++)
    {
        a[i]=(float)i;
        b[i]= -(float)i;
    }

	//copy data from host memory to device memory
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for ad\n%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    //printf("\nAfter HostToDevice Memcpy for bd\n%s\n",hipGetErrorString(hipGetLastError()));
	
    //calculate execution configuration
    if (N > 512)
    {
        No_of_threads = 512;
        No_of_blocks = (N / 512) + (((N % 512) == 0) ? 0 : 1);
       
    }
    else 
    {
        No_of_threads = N;
        No_of_blocks = 1;
    }
    dim3 block (No_of_threads, 1, 1);
    dim3 grid (No_of_blocks, 1, 1);
    
    //GPU timer code
    float time;
    hipEvent_t start,stop;			
	hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventRecord(start,0);
	
    //launch kernel with only one thread
    add_vectors<<< grid, block >>>(ad, bd, N);
	
    hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);			//time taken in kernel call calculated
	hipEventDestroy(start);
	hipEventDestroy(stop);

    //copy back the results
	hipMemcpy(a,ad,size,hipMemcpyDeviceToHost);
	//printf("\nAfter DeviceToHost Memcpy for a\n%s\n",hipGetErrorString(hipGetLastError()));
	
	//print the results
	printf("\nAddition of above two VECTORS on GPU evaluates to = \n");
    for (i = 0; i < N; i++)
        printf("%f\n", a[i]);                       //if correctly evaluated, all values will be 0
    printf("\n\nTime taken is %f (ms)\n",time);
    
    //deallocate host and device memories
    hipFree(ad); hipFree(bd);
	free(a);free(b);

	_getch();
    return 1;
}